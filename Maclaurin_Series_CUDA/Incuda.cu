
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void ln_function(float y, int n, float* result){
    float x = y -1.0f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        int sign = 1;
        if(idx%2 != 0){
            sign = -1;
        }
        result[idx]= sign * powf(x, idx+1) / (idx+1); 
    }
}


int main(int argc, char** argv){

    if(argc != 3){
        printf("Usage ./Incuda <num_terms> <values>\n");
        return 0;
    }

    int numTerms = atoi(argv[1]);
    float value = atof(argv[2]);

    if (numTerms <= 0 || value<= 1 || value>2){
        printf("Invalid input\n");
        printf("%d, %f", numTerms, value);
        return 1;
    }

    int N = numTerms;

    float *c;
    c = (float*)malloc(sizeof(float)*N);
   
    float *dc;
    hipMalloc((void**) &dc, sizeof(float)*N);

    //vector add
    ln_function<<<1, N>>>(value, numTerms, dc);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    //cudaDeviceSynchronize();
    hipDeviceSynchronize();

    //transfer data from device to host
    hipMemcpy(c, dc, sizeof(float)*N, hipMemcpyDeviceToHost);
    float result = 0.0f;
    for(int i = 0; i < N; i++){
        result += c[i];
        
        //printf("Term %d: %f\n", i, c[i]);
    }

    printf("%f\n", result);

    free(c); 
    hipFree(dc); 

    return 0;
}